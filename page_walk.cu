/***********************************************************************************
COMS 4824 Architecture Final Project
Implementing Page Walk runing on the GPU using CUDA

Copyright (c) 2017 Columbia University. 
All rights reserved.

Note: Needs compute capability >= 2.0, so compile with:
Usage:
    nvcc page_walk1.cu -arch=compute_20 -code=sm_20,compute_20 -o page_walk1.out
    ./page_walk1.out -n 4000 2 2 2 2 2 
************************************************************************************/


#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <string.h>
#include <errno.h>
#include <assert.h>
#include <sys/time.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <stdint.h>

#define BLOCK_D1 512
#define BLOCK_D2 1
#define BLOCK_D3 1
#define MAX_LEVELS 20

#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)


static int max_table;

struct trans_thread {
    void **curr_table;
    int offset[MAX_LEVELS];
    int curr;
    int max;
};

__host__ __device__ int translate_cpu(struct trans_thread *trans) {
    //void **c = trans->curr_table;
    while(trans->curr < trans->max-1) {
        trans->curr_table = (void **) trans->curr_table[trans->offset[trans->curr]];
        trans->curr++;
    }
    return 0;
    //(intptr_t) *(trans->curr_table + trans->offset[trans->curr]); // ((void *) trans->curr_table + trans->offset[trans->max-1]);
}

// CUDA kernel: gpu_run_time<<<gridSize, blockSize>>>(d_new_threads, total_addresses);
__global__ void gpu_run_time(struct trans_thread *trans, int addresses) {
    // note that this assumes no third dimension to the grid
    // id of the block
    int myblock = blockIdx.x + blockIdx.y * gridDim.x;
    // size of each block (within grid of blocks)
    int blocksize = blockDim.x * blockDim.y * blockDim.z;
    // id of thread in a given block
    int subthread = threadIdx.z*(blockDim.x * blockDim.y) + threadIdx.y*blockDim.x + threadIdx.x;
    // assign overall id/index of the thread
    int idx = myblock * blocksize + subthread;


        if(idx < addresses) {

            //printf("Hello world! My block index is (%d,%d) [Grid dims=(%d,%d)], 3D-thread index within block=(%d,%d,%d) => \
       thread index=%d\n", blockIdx.x, blockIdx.y, gridDim.x, gridDim.y, threadIdx.x, threadIdx.y, threadIdx.z, idx);
            //translate_cpu(&trans[idx]);
            /*while((*(trans+idx)).curr < (*(trans+idx)).max-1) {
                (*(trans+idx)).curr_table = (void **) (*(trans+idx)).curr_table[(*(trans+idx)).offset[(*(trans+idx)).curr]];
                (*(trans+idx)).curr++;
            }*/
            //translate_cpu(&trans[idx]);
            printf("Hello world!.\n");
        }
}


// CPU analog for speed comparison
float cpu_run_time(struct trans_thread *trans, int addresses) {
    //struct timeval start_time, stop_time;
    
    //gettimeofday(&start_time, NULL);

    for(int i = 0; i < addresses; i++) {
        //translate_cpu(&trans[i]);
        printf("Hello world!.\n");
    }

    //gettimeofday(&stop_time, NULL);

    //long time_diff = (stop_time.tv_usec)-(start_time.tv_usec);  
    return 0;
}

/* --------------------------- host code ------------------------------*/
double read_timer() {

    struct timeval end;
    gettimeofday( &end, NULL );
    return end.tv_sec+1.e-6*end.tv_usec;
}

int construct_table(void *table, int *levels, int num_levels) {
    int i, j, level_size = 1;
        void **table_ptr = (void **) table,
        **level_ptr;

    // set intermediate addresses of table
    for(i = 0; i < num_levels-1; i++)
    {
        level_size *= levels[i];

        // hideous but best way I could find to get next level
        //level_ptr = (void **) (table + (level_size)+ table_ptr);
        level_ptr = (void **) ((intptr_t)table + level_size+ ((intptr_t )table_ptr - (intptr_t )table));
        // helpufl check: fprintf(stderr, "level_size: %d, level_ptr: %d, table_ptr: %d\n", level_size, (level_ptr- (void **) table) / sizeof(void *), table_ptr -  (void **) table);

        for(j = 0; j < level_size; j++) {
            table_ptr[j] = level_ptr + ((j)*levels[i+1]);
        }   

        table_ptr += level_size;
    }
    assert((intptr_t )table_ptr - (intptr_t )table < max_table);


    // set last level of page table to garbage; for our purposes
    // it doesn't matter
    /*for(i = 0; i < level_size * levels[num_levels-1]; i++) {
        *table_ptr = (void *) i;
        table_ptr++;
    }*/


    // assert((intptr_t )table_ptr - (intptr_t )table == max_table);
    // return number of entries at the lowest level of the
    // page table
    return levels[num_levels-1] * level_size;
}

struct trans_thread *gen_addresses(int num_addr, int levels, int *level_sizes, void **pgd)
{
    int i,j;
    struct trans_thread *new_threads = (struct trans_thread *)malloc(sizeof(struct trans_thread) * num_addr);
    if (!new_threads){ 
        fprintf(stderr, "malloc failed: %d\n", strerror(errno));
        exit(1);
    }

    for(i = 0; i < num_addr; i++)
    {
        new_threads[i].curr_table = pgd;
        new_threads[i].max = levels;
        new_threads[i].curr = 0;

        for(j = 0; j < levels; j++) {
            new_threads[i].offset[j] = 
                rand() % level_sizes[j];
        }

    }

    return new_threads;
}

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv) {

    //void **pg_table; //host page table
    void **d_pg_table; //device page table

    int i, j, table_size = 0, level_size = 1, 
        total_addresses, table_lowest_addresses, 
        levels = argc-3;

    int level_sizes[levels];
    //struct trans_thread *sample;
    struct trans_thread *h_new_threads;
    struct trans_thread *d_new_threads;

    hipError_t cudaStat;


    printf("===============================================================================\n");
    srand(time(NULL));
    
    // get number of pointers in contiguous page table
    for(i = 1, j =0; i < argc; i++) {
        if ( !strcmp(argv[i], "-n")) {
            total_addresses = atoi(argv[++i]);
            continue;
        }
        level_size *=  atoi(argv[i]);
        level_sizes[j++] = atoi(argv[i]);
        table_size += level_size;
    }

    // fixed block dimensions (1024x1x1 threads)
    const dim3 blockSize(BLOCK_D1, BLOCK_D2, BLOCK_D3);
    // determine number of blocks we need for a given problem size
    int tmp = ceil(pow(total_addresses/BLOCK_D1, 0.5));
    printf("Grid dimension is %i x %i\n", tmp, tmp);
    dim3 gridSize(tmp, tmp, 1);

    int nthreads = BLOCK_D1*BLOCK_D2*BLOCK_D3*tmp*tmp;
    if (nthreads < total_addresses){
        printf("\n============ NOT ENOUGH THREADS TO COVER total addresses=%d ===============\n\n",total_addresses);
    } else {
        printf("Launching %d threads (total_addresses=%d)\n", nthreads, total_addresses);
    }

    // allocate host memory
    max_table = table_size * sizeof(void *); //total size of page table
    void **pg_table = (void **) malloc(sizeof(void *) * table_size);

    if (!pg_table) {
        fprintf(stderr, "host memory allocation failed: %d\n", strerror(errno));
        exit(1);
    }
    else {
        printf ("host memory allocation succeeded.\n");
    }

    // allocate device memory
    cudaStat = hipMalloc(&d_pg_table, sizeof(void *) * table_size);
    if(cudaStat != hipSuccess) {
      printf ("device memory allocation failed.\n");
      return EXIT_FAILURE;
    }
    else {
        printf ("device memory allocation succeeded.\n");
    }

    //number of entries at the lowest level of the page table
    //number of translatable addresses
    printf ("now construct the page table on the host.\n");
    table_lowest_addresses = construct_table(pg_table, level_sizes, levels);

    fprintf(stderr, "number of translatable addresses: %d\n", table_lowest_addresses);
    fprintf(stderr, "total size of page table: %d\n", max_table);

    hipDeviceSynchronize();
    double tInit = read_timer();

    // copy input data to the GPU
    cudaStat = hipMemcpy(d_pg_table, pg_table, sizeof(void *) * table_size, hipMemcpyHostToDevice);

    printf("Memory Copy for page table from Host to Device");
    if (cudaStat != hipSuccess){
      printf("failed.\n");
      return EXIT_FAILURE;
    } else {
      printf("successful.\n");
    }
    hipDeviceSynchronize();
    double tTransferToGPU_pgtable = read_timer();

    h_new_threads = gen_addresses(total_addresses, levels, level_sizes, pg_table);
    hipDeviceSynchronize();
    double tInit2 = read_timer();

    //Copy the struct to device memory
    cudaStat = hipMalloc( (void**) &d_new_threads, sizeof(struct trans_thread) * total_addresses) ;
    if (cudaStat != hipSuccess){
      printf("device memory allocation for d_new_threads failed.\n");
      return EXIT_FAILURE;
    } else {
      printf("device memory allocation for d_new_threads succeeded.\n");
    }

    cudaStat = hipMemcpy( d_new_threads, h_new_threads, sizeof(struct trans_thread) * total_addresses, hipMemcpyHostToDevice);
    printf("Memory Copy h_new_threads from Host to Device");
    if (cudaStat != hipSuccess){
      printf("failed.\n");
      return EXIT_FAILURE;
    } else {
      printf(" successful.\n");
    }
    cudaCheckErrors("hipMemcpy h_new_threads fail");

    hipDeviceSynchronize();
    double tTransferToGPU_threads = read_timer();

    gpu_run_time<<<gridSize, blockSize>>>(d_new_threads, total_addresses);
    hipDeviceSynchronize();
    double gpu_time = read_timer();

    hipError_t cudaerr = hipDeviceSynchronize();
    if (cudaerr){
        printf("kernel launch failed with error \"%s\".\n",
               hipGetErrorString(cudaerr));
    } else {
        printf("kernel launch success!\n");
    }

    
    printf("GPU done!\n");

    /*cudaStat = cudaMemcpy( h_new_threads, d_new_threads, sizeof(struct trans_thread) * total_addresses, cudaMemcpyDeviceToHost);
    //cudaMemcpy(cpu_vals, gpu_vals, N, cudaMemcpyDeviceToHost);
    printf("Memory Copy from Device to Host ");
    if (cudaStat){
      printf("failed.\n");
    } else {
      printf("successful.\n");
    }
    cudaCheckErrors("cudaMemcpy fail");

    cudaDeviceSynchronize();
    double tTransferFromGPU = read_timer();*/


    printf("now do calculation on CPU for comparison!\n");
    cpu_run_time(h_new_threads, total_addresses);   
    double cpu_time = read_timer();
    
    fprintf(stderr, "The CPU took %lu microseconds to compute %d addresses. ""For a table of depth %d.\n", 
            cpu_time - gpu_time , total_addresses, levels);

    printf("Timing results for n = %d\n", total_addresses);
    printf("page table Transfer to GPU time: %f\n", tTransferToGPU_pgtable - tInit);
    printf("threads Transfer to GPU time: %f\n", tTransferToGPU_threads - tInit2);
    printf("Calculation time (GPU): %f\n", gpu_time - tTransferToGPU_threads);
    //printf("Transfer from GPU time: %f\n", tTransferFromGPU - gpu_time);
    printf("Calculation time (CPU): %f\n", cpu_time - gpu_time);
  

    printf("Freeing memory...\n");
    printf("====================================================\n");
    free(pg_table);
    free(h_new_threads);
    hipFree(d_pg_table);
    hipFree(d_new_threads);
    return 0;
}
